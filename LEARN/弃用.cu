#include "hip/hip_runtime.h"
// 计算一层 CCT 在 p 点产生的磁场。注意磁场还要再乘电流 和 μ0/4π (=1e-7)
// ksi_deg0 是计算的起点。wind_num 是匝数。
// phi0、k_tilt_angles、a、ch_eta0、sh_eta0 见 point_cct 函数，p 点为需要计算磁场的点，m_ret 是返回的磁场。
__device__ void // 不再使用，粒度太大
magnet_cct(int ksi_deg0, int wind_num, float phi0, float *k_tilt_angles, float a, float ch_eta0, float sh_eta0,
           float *p, float *m_ret);


__device__ void
magnet_cct_parallel(int ksi_deg0, int wind_num, float phi0, float *k_tilt_angles, float a, float ch_eta0, float sh_eta0,
                    float current, float *p, /*__shared__*/ float *m_ret);


__device__ void // 并行度低!! 仅仅用于测试
magnet_cct(int ksi_deg0, int wind_num, float phi0, float *k_tilt_angles, float a, float ch_eta0, float sh_eta0,
           float *p, float *m_ret) {
    int wi;
    float m_pre_wind[3];
    vct_zero(m_ret); // m = 0,0,0

    for (wi = 0; wi < wind_num; wi++) {
        dB_cct_wind(ksi_deg0, phi0, k_tilt_angles, a, ch_eta0, sh_eta0, p, m_pre_wind);
        ksi_deg0 += 360;
        vct_add_local(m_ret, m_pre_wind);
    }
}

// 并发执行。这个方法和非同步的 magnet_cct 最大的不同是，返回值 m_ret 是真实磁场，不需要 current * 1e-7f，因此需要传入 CCT 电流 current
// 注意 m_ret 必须是 __shared__，其他参数意义见 magnet_cct。m_ret 不必置零，在此方法内完成
// 另外此方法本身就是同步方法，因为最后一行代码是 __syncthreads();，所以调用后无需执行额外的
__device__  void // 此函数粒度不够，不再使用
magnet_cct_parallel(int ksi_deg0, int wind_num, float phi0, float *k_tilt_angles, float a, float ch_eta0, float sh_eta0,
                    float current, float *p, /*__shared__*/ float *m_ret) {
    int wi = (int) threadIdx.x; // 块内计算，每块计算一个 CCT，wind_num ≤ threadIdx

    if (wi == 0) {
        vct_zero(m_ret);
    }

    // 对共享内存写其他线程看得到
    __syncthreads();

    float m_part[3];

    if (wi < wind_num) {
        dB_cct_wind(ksi_deg0 + 360 * wi, phi0, k_tilt_angles, a, ch_eta0, sh_eta0, p, m_part);
        vct_dot_a_v(current * 1e-7f, m_part);
    }

    atomicAdd(&m_ret[X], m_part[X]);
    atomicAdd(&m_ret[Y], m_part[Y]);
    atomicAdd(&m_ret[Z], m_part[Z]);

    // 对共享内存写所有线程完成
    __syncthreads();
}

// 2020年11月13日 测试通过
__global__ void test_magnet_cct()
{
    int ksi_deg0 = 0;
    int wind_num = 128;
    float bendingAngle = 67.5f;
    float bendingRad = bendingAngle / 180.0f * PI;
    float tiltAngles[TILE_ANGLE_LENGTH] = {30.f, 80.f, 90.f, 90.f};
    float bigR = 0.95f;
    float smallR = 83 * MM + 15 * MM * 2;
    float phi0 = bendingRad / (float) wind_num;
    float current = -9664.f;

    float a = sqrtf(bigR * bigR - smallR * smallR);
    float eta0 = 0.5f * logf((bigR + a) / (bigR - a));
    float ch_eta0 = coshf(eta0);
    float sh_eta0 = sinhf(eta0);

    float k[TILE_ANGLE_LENGTH];

    k[0] = (1.0f / tanf(tiltAngles[0] / 180.0f * PI)) / ((float) (0 + 1) * sh_eta0);
    k[1] = (1.0f / tanf(tiltAngles[1] / 180.0f * PI)) / ((float) (1 + 1) * sh_eta0);
    k[2] = 0.0;
    k[3] = 0.0;

    float p[3] = {0, 0, 0};
    float m[3] = {0, 0, 0};

    magnet_cct(ksi_deg0, wind_num, phi0, k, a, ch_eta0, sh_eta0, p, m);
//    magnet_cct_parallel(ksi_deg0, wind_num, phi0, k, a, ch_eta0, sh_eta0, p, m);

    printf("device -- %f, %f, %f\n", m[X], m[Y], m[Z]);

    vct_dot_a_v(current * 1e-7f, m);

    // Java --  [0.0031436355039083964, -0.00470478301086915, 0.00888627084434009]
    // device -- 0.003144, -0.004705, 0.008886
    printf("device -- %f, %f, %f\n", m[X], m[Y], m[Z]);
}

// 并发计算每匝线圈磁场，测试成功 2020年11月14日
__global__ void test_magnet_cct_parallel() {
    unsigned int tid_block = threadIdx.x;

    int ksi_deg0 = 0;
    int wind_num = 128;
    float bendingAngle = 67.5f;
    float bendingRad = bendingAngle / 180.0f * PI;
    float tiltAngles[TILE_ANGLE_LENGTH] = {30.f, 80.f, 90.f, 90.f};
    float bigR = 0.95f;
    float smallR = 83 * MM + 15 * MM * 2;
    float phi0 = bendingRad / (float) wind_num;
    float current = -9664.f;

    float a = sqrtf(bigR * bigR - smallR * smallR);
    float eta0 = 0.5f * logf((bigR + a) / (bigR - a));
    float ch_eta0 = coshf(eta0);
    float sh_eta0 = sinhf(eta0);

    float k[TILE_ANGLE_LENGTH];

    k[0] = (1.0f / tanf(tiltAngles[0] / 180.0f * PI)) / ((float) (0 + 1) * sh_eta0);
    k[1] = (1.0f / tanf(tiltAngles[1] / 180.0f * PI)) / ((float) (1 + 1) * sh_eta0);
    k[2] = 0.0;
    k[3] = 0.0;

    float p[3] = {0, 0, 0};
    __shared__ float m[3];

    magnet_cct_parallel(ksi_deg0, wind_num, phi0, k, a, ch_eta0, sh_eta0, current, p, m);

    // 少一次同步
//    if (tid_block == 0) {
//        vct_dot_a_v(current * 1e-7f, m);
//    }
//    __syncthreads();

    // Java --  [0.0031436355039083964, -0.00470478301086915, 0.00888627084434009]
    if (tid_block == 0) {
        //device -- 0.003144, -0.004705, 0.008886
        printf("device -- %f, %f, %f\n", m[X], m[Y], m[Z]);
    }

}

// 2020年11月13日 测试成功
__global__ void test_particle_run() {
    int ksi_deg0 = 0;
    int wind_num = 128;
    float bendingAngle = 67.5f;
    float bendingRad = bendingAngle / 180.0f * PI;
    float tiltAngles[TILE_ANGLE_LENGTH] = {30.f, 80.f, 90.f, 90.f};
    float bigR = 0.95f;
    float smallR = 83 * MM + 15 * MM * 2;
    float phi0 = bendingRad / (float) wind_num;
    float current = -9664.f;

    float a = sqrtf(bigR * bigR - smallR * smallR);
    float eta0 = 0.5f * logf((bigR + a) / (bigR - a));
    float ch_eta0 = coshf(eta0);
    float sh_eta0 = sinhf(eta0);

    float k[TILE_ANGLE_LENGTH];

    k[0] = (1.0f / tanf(tiltAngles[0] / 180.0f * PI)) / ((float) (0 + 1) * sh_eta0);
    k[1] = (1.0f / tanf(tiltAngles[1] / 180.0f * PI)) / ((float) (1 + 1) * sh_eta0);
    k[2] = 0.0;
    k[3] = 0.0;

    float p[3] = {0, 0, 0};
    float v[3] = {0.0, 0.0, 1.839551780274753E8};
    float rm = 2.1182873748205775E-27;
    float speed = 1.839551780274753E8;
    float m[3];

    float distance = 0.0f;
    float LENGTH = 1.0f;

    while (distance < LENGTH) {
        // 求磁场
        magnet_cct(ksi_deg0, wind_num, phi0, k, a, ch_eta0, sh_eta0, p, m);
//        magnet_cct_parallel(ksi_deg0, wind_num, phi0, k, a, ch_eta0, sh_eta0, p, m);
        vct_dot_a_v(current * 1e-7f, m);
        // run
        particle_run_step(m, p, v, rm, speed);
        distance += STEP_RUN;
    }
    vct_print(p);
    vct_print(v);
    // last print
    //0.000170, 0.001429, 1.000991
    //-1064.316528, 519703.968750, 183955184.000000
    // Java :
    // position=[1.696032050934739E-4, 0.0014294050422786041, 1.0009986107410314]
    // velocity=[-1064.0616149611867, 519703.2016243721, 1.8395444471527618E8]
}

// 2020年11月14日 测试通过
__global__ void test_particle_run_parallel() {
    unsigned int tid_block = threadIdx.x;

    int ksi_deg0 = 0;
    int wind_num = 128;
    float bendingAngle = 67.5f;
    float bendingRad = bendingAngle / 180.0f * PI;
    float tiltAngles[TILE_ANGLE_LENGTH] = {30.f, 80.f, 90.f, 90.f};
    float bigR = 0.95f;
    float smallR = 83 * MM + 15 * MM * 2;
    float phi0 = bendingRad / (float) wind_num;
    float current = -9664.f;

    float a = sqrtf(bigR * bigR - smallR * smallR);
    float eta0 = 0.5f * logf((bigR + a) / (bigR - a));
    float ch_eta0 = coshf(eta0);
    float sh_eta0 = sinhf(eta0);

    float k[TILE_ANGLE_LENGTH];

    k[0] = (1.0f / tanf(tiltAngles[0] / 180.0f * PI)) / ((float) (0 + 1) * sh_eta0);
    k[1] = (1.0f / tanf(tiltAngles[1] / 180.0f * PI)) / ((float) (1 + 1) * sh_eta0);
    k[2] = 0.0;
    k[3] = 0.0;

    float p[3] = {0, 0, 0};
    float v[3] = {0.0, 0.0, 1.839551780274753E8};
    float rm = 2.1182873748205775E-27;
    float speed = 1.839551780274753E8;

    // 必须是共享变量
    __shared__ float m[3];

    float distance = 0.0f;
    float LENGTH = 100.0f;

    while (distance < LENGTH) {
        // 求磁场。无需同步
        magnet_cct_parallel(ksi_deg0, wind_num, phi0, k, a, ch_eta0, sh_eta0, current, p, m);
        // run
        particle_run_step(m, p, v, rm, speed);

        distance += STEP_RUN;
    }

    if (tid_block == 0) {
        vct_print(p);
        vct_print(v);
    }
    // 100 m
    // Java position=[-0.002617523327502486, 0.3855863500772864, 100.04325856386859]
    // Java velocity=[-3304.0625271094736, 713319.4063558772, 1.8395379588762134E8]
    // -0.002617, 0.385373, 100.000671
    // -3303.234375, 713905.625000, 183955184.000000


    // 1m
    // last print
    //0.000170, 0.001429, 1.000991
    //-1064.316528, 519703.968750, 183955184.000000
    // Java :
    // position=[1.696032050934739E-4, 0.0014294050422786041, 1.0009986107410314]
    // velocity=[-1064.0616149611867, 519703.2016243721, 1.8395444471527618E8]
}

// 测试相对位置 2020年11月14日 测试通过
__global__ void test_parallel_relative_position() {
    unsigned int tid_block = threadIdx.x;

    int ksi_deg0 = 0;
    int wind_num = 128;
    float bendingAngle = 67.5f;
    float bendingRad = bendingAngle / 180.0f * PI;
    float tiltAngles[TILE_ANGLE_LENGTH] = {30.f, 80.f, 90.f, 90.f};
    float bigR = 0.95f;
    float smallR = 83.f * MM + 15.f * MM * 2.f;
    float phi0 = bendingRad / (float) wind_num;
    float current = -9664.f;

    float a = sqrtf(bigR * bigR - smallR * smallR);
    float eta0 = 0.5f * logf((bigR + a) / (bigR - a));
    float ch_eta0 = coshf(eta0);
    float sh_eta0 = sinhf(eta0);

    float k[TILE_ANGLE_LENGTH];

    k[0] = (1.0f / tanf(tiltAngles[0] / 180.0f * PI)) / ((float) (0 + 1) * sh_eta0);
    k[1] = (1.0f / tanf(tiltAngles[1] / 180.0f * PI)) / ((float) (1 + 1) * sh_eta0);
    k[2] = 0.0f;
    k[3] = 0.0f;

    float p[3] = {5.0085219608773155, 2.951165121396268, 0.0};
    float v[3] = {0.0f, 0.0f, 1.839551780274753E8f};
    float rm = 2.1182873748205775E-27f;
    float speed = 1.839551780274753E8f;

    float pr[3]; // 相对点
    float m[3]; // 绝对磁场

    __shared__ float mr[3];
    cct345_1_relative_point(p, pr);
    magnet_cct_parallel(ksi_deg0, wind_num, phi0, k, a, ch_eta0, sh_eta0, current, pr, mr);
    cct345_1_absolute_m(mr, m);

    if (tid_block == 0) {
        vct_print(p);
        vct_print(v);
        vct_print(m);
    }

}

// 测试相对位置下粒子运动 2020年11月14日 测试通过
__global__ void test_parallel_relative_position_particle_run() {
    unsigned int tid_block = threadIdx.x;

    int ksi_deg0 = 0;
    int wind_num = 128;
    float bendingAngle = 67.5f;
    float bendingRad = bendingAngle / 180.0f * PI;
    float tiltAngles[TILE_ANGLE_LENGTH] = {30.f, 80.f, 90.f, 90.f};
    float bigR = 0.95f;
    float smallR = 83.f * MM + 15.f * MM * 2.f;
    float phi0 = bendingRad / (float) wind_num;
    float current = -9664.f;

    float a = sqrtf(bigR * bigR - smallR * smallR);
    float eta0 = 0.5f * logf((bigR + a) / (bigR - a));
    float ch_eta0 = coshf(eta0);
    float sh_eta0 = sinhf(eta0);

    float k[TILE_ANGLE_LENGTH];

    k[0] = (1.0f / tanf(tiltAngles[0] / 180.0f * PI)) / ((float) (0 + 1) * sh_eta0);
    k[1] = (1.0f / tanf(tiltAngles[1] / 180.0f * PI)) / ((float) (1 + 1) * sh_eta0);
    k[2] = 0.0f;
    k[3] = 0.0f;

    float p[3] = {3.5121278119986163, 1.45477097251757, 0.0};
    float v[3] = {1.300759538176064E8, 1.3007595381760634E8, 0.0};
    float rm = 2.1182873748205775E-27f;
    float speed = 1.839551780274753E8f;

    float pr[3]; // 相对点
    float m[3]; // 绝对磁场

    __shared__ float mr[3];

    float distance = 0.0f;
    float LENGTH = 2.0f;

    while (distance < LENGTH) {
        // 相对点
        cct345_1_relative_point(p, pr);
        // 相对磁场
        magnet_cct_parallel(ksi_deg0, wind_num, phi0, k, a, ch_eta0, sh_eta0, current, pr, mr);
        // 绝对磁场
        cct345_1_absolute_m(mr, m);

        // run
        particle_run_step(m, p, v, rm, speed);

        distance += STEP_RUN;
    }


    if (tid_block == 0) {
        vct_print(p);
        vct_print(v);
    }

}

// 计算整个二极CCT产生的磁场。详见 test_magnet_bicct345_parallel_fill_data 2020年11月15日 测试通过
__global__ void test_magnet_bicct345_parallel(float *data) {
    unsigned int tid = threadIdx.x;

    float p[3] = {0, 0, 0};
    float m_per_wind[3];

    __shared__ float m_total[3];
    if (tid == 0) {
        vct_zero(m_total);
    }
    __syncthreads();

    // 外层
    // Java -0.004067504914360193, 0.006087451294501636, 0.011784791740462989
    // ret -0.004068, 0.006087, 0.011785
//    if (tid > 127 && tid < 256) { // 两层 bi cct

    // 内层
    // Java 0.0031436355039083964, -0.00470478301086915, 0.00888627084434009
    // ret 0.003144, -0.004705, 0.008886
//    if (tid < 128) { // 两层 bi cct

    // Java -9.238694104517966E-4, 0.0013826682836324865, 0.020671062584803078
    // ret -0.000924, 0.001383, 0.020671
    if (tid < 256) {
        dB_cct_wind(
                *((int *) (data + tid * NUMBER_OF_VARIABLES_PER_CCT + 0)), // ksi_deg0
                *(data + tid * NUMBER_OF_VARIABLES_PER_CCT + 10), // start_phi
                *(data + tid * NUMBER_OF_VARIABLES_PER_CCT + 1), // phi0
                data + tid * NUMBER_OF_VARIABLES_PER_CCT + 2, // k
                *(data + tid * NUMBER_OF_VARIABLES_PER_CCT + 6), // a
                *(data + tid * NUMBER_OF_VARIABLES_PER_CCT + 7), // ch_eta0
                *(data + tid * NUMBER_OF_VARIABLES_PER_CCT + 8), // sh_eta0
                p, m_per_wind
        );

        vct_dot_a_v((*(data + tid * NUMBER_OF_VARIABLES_PER_CCT + 9)) * 1e-7f, m_per_wind);

        atomicAdd(&m_total[X], m_per_wind[X]);
        atomicAdd(&m_total[Y], m_per_wind[Y]);
        atomicAdd(&m_total[Z], m_per_wind[Z]);
    }

    __syncthreads();

    if (tid == 0) {
        vct_print(m_total);
    }
}

// 计算整个 AG-CCT 产出的磁场
// 计算整个二极CCT产生的磁场。详见 test_magnet_bicct345_parallel_fill_data 2020年11月15日 测试通过
__global__ void test_magnet_agcct345_parallel(float *data) {
    unsigned int tid = threadIdx.x;

    // Java -0.008153130729662327, 0.12075963610910645, -2.128470062057155
    float p[3] = {0.95, 0.1, 0}; // -0.008154, 0.120739, -2.128648
    float m_per_wind[3];

    __shared__ float m_total[3];
    if (tid == 0) {
        vct_zero(m_total);
    }
    __syncthreads();


    if (tid < CCT_TOTAL_WIND_NUM) {
        dB_cct_wind(
                *((int *) (data + tid * NUMBER_OF_VARIABLES_PER_CCT + 0)), // ksi_deg0
                *(data + tid * NUMBER_OF_VARIABLES_PER_CCT + 10), // start_phi
                *(data + tid * NUMBER_OF_VARIABLES_PER_CCT + 1), // phi0
                data + tid * NUMBER_OF_VARIABLES_PER_CCT + 2, // k
                *(data + tid * NUMBER_OF_VARIABLES_PER_CCT + 6), // a
                *(data + tid * NUMBER_OF_VARIABLES_PER_CCT + 7), // ch_eta0
                *(data + tid * NUMBER_OF_VARIABLES_PER_CCT + 8), // sh_eta0
                p, m_per_wind
        );

        vct_dot_a_v((*(data + tid * NUMBER_OF_VARIABLES_PER_CCT + 9)) * 1e-7f, m_per_wind);

        atomicAdd(&m_total[X], m_per_wind[X]);
        atomicAdd(&m_total[Y], m_per_wind[Y]);
        atomicAdd(&m_total[Z], m_per_wind[Z]);
    }

    __syncthreads();

    if (tid == 0) {
        vct_print(m_total);
    }
}

__global__ void test_particle_run_parallel(float *data) {
    unsigned int tid = threadIdx.x;

    float p[3] = {3.5121278119986163, 1.45477097251757, 0.0};
    float v[3] = {1.2326128074269669E8, 1.2326128074269663E8, 0.0};
    float rm = 2.0558942080656965E-27;
    float speed = 1.7431777494179922E8;

    float distance = 0.0f;
    float LENGTH = 3.433224382841364f;

    float pr_cct[2][3]; // 相对点
    float m[2][3]; // 绝对磁场

    __shared__ float mr[2][3]; // 相对磁场

    while (distance < LENGTH) {
        // 相对点
        cct345_1_relative_point(p, pr_cct[0]);
        // 磁场
        magnet_at_cct345_1(data, pr_cct[0], mr[0]);
        // 绝对磁场
        cct345_1_absolute_m(mr[0], m[0]);
        // 粒子运动
        particle_run_step(m[0], p, v, rm, speed);

        distance += STEP_RUN;
    }

    if (tid == 0) {
        // java
        // {position=[3.5121278119986163, 1.45477097251757, 0.0],
        // velocity=[1.2326128074269669E8, 1.2326128074269663E8, 0.0], distance=0.0}
        // {position=[6.225937108498759, 3.0639867524971423, -0.001508755653133722],
        // velocity=[1.595159970266312E8, -7.054897849029969E7, -279805.24817167915], distance=3.433224382841577}

        // cuda
        // 6.226668, 3.063664, -0.001510
        //159516464.000000, -70547552.000000, -279971.218750

        // diff
        // ([-7.30891501e-04,  3.22752497e-04,  1.24434687e-06]) 单位 m
        // ([-0.7308915 ,  0.3227525 ,  0.00124435]) 单位 mm
        vct_print(p);
        vct_print(v);
        printf("%f\n", distance); // 3.433933
    }
}

// 测试 CCT345_2 磁场，成了
__global__ void test_cct345_2_magnet(float *data) {
    unsigned int tid = threadIdx.x;

    float p[3] = {7.157070170396251, 2.366002833170567, 0.0};

    float pr_cct[2][3]; // 相对点
    float m[2][3]; // 绝对磁场

    __shared__ float mr[2][3]; // 相对磁场

    cct345_2_relative_point(p, pr_cct[1]);

    magnet_at_cct345_1(data, pr_cct[1], mr[1]);

    cct345_2_absolute_m(mr[1], m[1]);

    if (tid == 0) {
        // Java
        // -0.00979169237915073, 0.027465786414956783, 2.403755237120381
        // cuda
        // -0.009778, 0.027427, 2.403643
        vct_print(m[1]);
    }
}

__global__ void test_cct345_1_and_2_magnet(float *data) {
    unsigned int tid = threadIdx.x;

    float p[3] = {6.336646882102519, 3.035807472522424, 0.0};

    float pr_cct[2][3]; // 相对点
    float m[2][3]; // 绝对磁场

    __shared__ float mr[2][3]; // 相对磁场

    cct345_1_relative_point(p, pr_cct[0]);
    cct345_2_relative_point(p, pr_cct[1]);

    magnet_at_cct345_1_and_2(data, pr_cct, mr);

    cct345_1_absolute_m(mr[0], m[0]);
    cct345_2_absolute_m(mr[1], m[1]);

    if (tid == 0) {
        vct_print(m[1]);
        vct_print(m[0]);

        vct_add_local(m[0], m[1]);
        // java
        //
        // cuda
        //
        vct_print(m[0]);
    }
}

// 测试 qs3 ，测试通过 2020年11月16日
__global__ void test_magnet_at_qs3(float *h_data) {
    float p0[3] = {6.235815085670027, 3.0775733701232055, 0.0};
    float m0[3];

    magnet_at_qs3(-7.3733f, -45.31f * 2, p0, m0);
    if (threadIdx.x == 0) {
        vct_print(m0);
    }

    //-------------------

    // new point
    float p1[3] = {6.239641919993678, 3.0868121654483183, 0.0};
    float m1[3];

    magnet_at_qs3(200, 0.0 * 2, p1, m1);
    if (threadIdx.x == 0) {
        // -0.000000, -0.000000, 2.000050
        // 0.0, 0.0, 2.000000000000078
        vct_print(m1);
    }

    // new point
    float p2[3] = {6.235815085670027, 3.0775733701232055, 0.01};
    float m2[3];

    magnet_at_qs3(200, 0.0 * 2, p2, m2);
    if (threadIdx.x == 0) {
        // 0.765367, 1.847759, 0.000000
        // 0.7653668647301795, 1.8477590650225735, -0.0
        vct_print(m2);
    }

    // new point
    float p3[3] = {6.235815085670027, 3.0775733701232055, 0.01};
    float m3[3];

    magnet_at_qs3(0.0, 200 * 2, p3, m3);
    if (threadIdx.x == 0) {
        // 0.000000, 0.000000, -0.020000
        // -0.0, -0.0, -0.02
        vct_print(m3);
    }

    // new point
    float p4[3] = {6.235815085670027, 3.0775733701232055, 0.01};
    float m4[3];

    magnet_at_qs3(0.0, 200 * 2, p4, m4);
    if (threadIdx.x == 0) {
        // 0.000000, 0.000000, -0.020000
        // 0.0, 0.0, 0.02000000000000156
        vct_print(m4);
    }

    // new point
    float p5[3] = {6.34985075343268, 3.0565930066029123, 0.07383321438269272};
    float m5[3];

    magnet_at_qs3(10, 10000 * 2, p5, m5);
    if (threadIdx.x == 0) {
        // 13.989565, 33.773796, -48.387268
        // 13.989622058936229, 33.77393530715767, -48.38722507873593
        vct_print(m5);
    }

}

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define MM 0.001f
#define DIM 3
#define PI 3.1415927f
#define X 0
#define Y 1
#define Z 2
#define Proton_Charge_Quantity 1.6021766208e-19f
#define Proton_Static_MassKg 1.672621898e-27f
#define Proton_Static_MassMeV 938.2720813f
#define Light_Speed 299792458.0f

// 默认 CCT 分段为 3 度，这是满足计算精度下最粗的分段
#define STEP_KSI 3

// 粒子运动步长，默认 1mm
#define STEP_RUN 0.001f

// 倾斜角几个，默认 4 个，即二级场、四极场、六极场、八级场，如果修个这个参数，需要修改方法 ksi_phi_fun 因为为了性能写死了
#define TILE_ANGLE_LENGTH 4

#define SIN_45 0.7071067811865476f
#define COS_45 0.7071067811865476f

// 机架移动
#define CCT345_1_MOVE_X 5.680273403004535f
#define CCT345_1_MOVE_Y 2.279413679269048f

// 每匝 CCT 需要的参数 9 个 起点 ksi0，匝弧度 phi0，k[0][1][2][3]，极角 a，ch_eta0，sh_eta0，电流current，起点 phi_start
#define NUMBER_OF_VARIABLES_PER_CCT 11

// 简单向量常量操作
__device__ __forceinline__ void vct_cross(float *a, float *b, float *ret);

__device__ __forceinline__ void vct_add_local(float *a_local, float *b);

__device__ __forceinline__ void vct_add(float *a, float *b, float *ret);

__device__ __forceinline__ void vct_sub(float *a, float *b, float *ret);

__device__ __forceinline__ void vct_dot_a_v(float a, float *v);

__device__ __forceinline__ void vct_dot_a_v_ret(float a, float *v, float *ret);

__device__ __forceinline__ void vct_copy(float *src, float *des);

__device__ __forceinline__ float vct_len(float *v);

__device__ __forceinline__ void vct_zero(float *v);

__device__ __forceinline__ void vct_print(float *v);

__device__ __forceinline__ float deg2rad(int deg); // 角度转弧度。本代码中，角度一定是整数。这个方法，以后可能要打表
__device__ __forceinline__ float sin_deg(int deg); // 三角函数，参数整数的角度。这个方法，以后可能要打表。--re. 2020年11月14日 打表意义不大
__device__ __forceinline__ float cos_deg(int deg); // 同上

// 磁场计算 注意，这里计算的不是电流元的磁场，还需要乘以 电流 和 μ0/4π (=1e-7)
__device__ void dB(float *p0, float *p1, float *p, float *ret);

// ksi phi 函数。phi0 即一匝线圈后，大半径转过的弧度。k_tilt_angles 是倾斜角系数 == cot(倾斜角[i])/(i+1)sinh(eta)
__device__ __forceinline__ float ksi_phi_fun(int ksi_deg, float phi_start, float phi0, float *k_tilt_angles);

// 计算 CCT 上 ksi_deg 处的点，存放在 p_ret 中。k_tilt_angles 的含义见 ksi_phi_fun，a 是极角。ch_eta0 = ch(eta0)，sh_eta0 = sh(eta0)
__device__ __forceinline__ void
point_cct(int ksi_deg, float phi_start, float phi0, float *k_tilt_angles, float a, float ch_eta0, float sh_eta0,
          float *p_ret);

// 计算一匝 CCT 线圈在 p 点产生的磁场，注意磁场还要再乘电流 和 μ0/4π (=1e-7)
// ksi_deg0 是计算的起点。phi0、k_tilt_angles、a、ch_eta0、sh_eta0 见 point_cct 函数，p 点为需要计算磁场的点，m_ret 是返回的磁场
__device__ void
dB_cct_wind(int ksi_deg0, float phi_start, float phi0, float *k_tilt_angles, float a, float ch_eta0, float sh_eta0,
            float *p,
            float *m_ret);

// 粒子走一步 m 磁场，p 位置，v 速度，rm 动质量，sp 速率。默认步长 STEP_RUN == 1mm
__device__  __forceinline__  void particle_run_step(float *m, float *p, float *v, float run_mass, float speed);

// 在 Java 的 CCT 建模中，我们移动的是 CCT，将 CCT 平移 / 旋转 到指定的位置，但是这么做开销很大
// 与其移动带有上万个点的 CCT 模型，不如移动只有 1 个点的粒子。 p 为绝对坐标点，pr 为相对于 cct345_1（后偏转段第一段 CCT）的点
// 因此此函数的使用方法为，首先已知绝对坐标下的粒子 p，利用此函数求相对点 pr，然后进行磁场计算，得到的磁场也仅仅是相对磁场，
// 再利用 cct345_1_absolute_m 把相对磁场转为绝对磁场
// 此函数中带有大量的魔数，如果修改了机架模型的长度 / 位置，必须做出调整
__device__ __forceinline__ void cct345_1_relative_point(float *p, float *pr);

// 函数意义见 cct345_1_relative_point
__device__ __forceinline__ void cct345_1_absolute_m(float *mr, float *m);


/***************** DEFINE **********************/

__device__ __forceinline__ void vct_cross(float *a, float *b, float *ret) {
    ret[X] = a[Y] * b[Z] - a[Z] * b[Y];
    ret[Y] = -a[X] * b[Z] + a[Z] * b[X];
    ret[Z] = a[X] * b[Y] - a[Y] * b[X];
}

__device__ __forceinline__ void vct_add_local(float *a_local, float *b) {
    a_local[X] += b[X];
    a_local[Y] += b[Y];
    a_local[Z] += b[Z];
}

__device__ __forceinline__ void vct_add(float *a, float *b, float *ret) {
    ret[X] = a[X] + b[X];
    ret[Y] = a[Y] + b[Y];
    ret[Z] = a[Z] + b[Z];
}

__device__ __forceinline__ void vct_dot_a_v(float a, float *v) {
    v[X] *= a;
    v[Y] *= a;
    v[Z] *= a;
}

__device__ __forceinline__ void vct_dot_a_v_ret(float a, float *v, float *ret) {
    ret[X] = v[X] * a;
    ret[Y] = v[Y] * a;
    ret[Z] = v[Z] * a;
}

__device__ __forceinline__ void vct_copy(float *src, float *des) {
    des[X] = src[X];
    des[Y] = src[Y];
    des[Z] = src[Z];
}

__device__ __forceinline__ float vct_len(float *v) {
    return sqrtf(v[X] * v[X] + v[Y] * v[Y] + v[Z] * v[Z]);
}

__device__ __forceinline__ void vct_zero(float *v) {
    v[X] = 0.0f;
    v[Y] = 0.0f;
    v[Z] = 0.0f;
}

__device__ __forceinline__ void vct_print(float *v) {
    printf("%f, %f, %f\n", v[X], v[Y], v[Z]);
}

__device__ __forceinline__ void vct_sub(float *a, float *b, float *ret) {
    ret[X] = a[X] - b[X];
    ret[Y] = a[Y] - b[Y];
    ret[Z] = a[Z] - b[Z];
}

__device__ __forceinline__ float deg2rad(int deg) {
    return ((float) deg) * PI / 180.0f;
}

__device__ __forceinline__ float sin_deg(int deg) {
    return __sinf(deg2rad(deg));
}

__device__ __forceinline__ float cos_deg(int deg) {
    return __cosf(deg2rad(deg));
}

__device__ void dB(float *p0, float *p1, float *p, float *ret) {
    float p01[DIM];
    float r[DIM];
    float rr;

    vct_sub(p1, p0, p01); // p01 = p1 - p0

    vct_add(p0, p1, r); // r = p0 + p1

    vct_dot_a_v(0.5f, r); // r = (p0 + p1)/2

    vct_sub(p, r, r); // r = p - r

    rr = vct_len(r); // rr = len(r)

    vct_cross(p01, r, ret); // ret = p01 x r

    rr = 1.0f / rr / rr / rr; // changed

    vct_dot_a_v(rr, ret); // rr . (p01 x r)
}

__device__ __forceinline__ float ksi_phi_fun(int ksi_deg, float phi_start, float phi0, float *k_tilt_angles) {
    // k 数组长度是 TILE_ANGLE_LENGTH，默认 4
    float ksi_rad = deg2rad(ksi_deg);

    return phi0 / (2.0f * PI) * ksi_rad +
           k_tilt_angles[0] * sin_deg(ksi_deg) +
           k_tilt_angles[1] * sin_deg(2 * ksi_deg) +
           k_tilt_angles[2] * sin_deg(3 * ksi_deg) +
           k_tilt_angles[3] * sin_deg(4 * ksi_deg) +
           phi_start;
}

__device__ __forceinline__ void
point_cct(int ksi_deg, float phi_start, float phi0, float *k_tilt_angles, float a, float ch_eta0, float sh_eta0,
          float *p_ret) {
    float phi = ksi_phi_fun(ksi_deg, phi_start, phi0, k_tilt_angles);
    float temp = a / (ch_eta0 - cos_deg(ksi_deg));

    p_ret[X] = temp * sh_eta0 * cosf(phi); // 太惨了，这个地方不能打表
    p_ret[Y] = temp * sh_eta0 * sinf(phi); // 太惨了，这个地方不能打表
    p_ret[Z] = temp * sin_deg(ksi_deg);
}

__device__ void
dB_cct_wind(int ksi_deg0, float phi_start, float phi0, float *k_tilt_angles, float a, float ch_eta0,
            float sh_eta0, float *p, float *m_ret) {
    int end_ksi_deg = ksi_deg0 + 360;
    float pre_point[3];
    float cur_point[3];
    float delta_B[3];

    point_cct(ksi_deg0, phi_start, phi0, k_tilt_angles, a, ch_eta0, sh_eta0, pre_point); // 起点

    vct_zero(m_ret); // m = 0,0,0

    while (ksi_deg0 < end_ksi_deg) {
        ksi_deg0 += STEP_KSI;

        point_cct(ksi_deg0, phi_start, phi0, k_tilt_angles, a, ch_eta0, sh_eta0, cur_point); // 下一个点

        dB(pre_point, cur_point, p, delta_B); // 计算磁场

        vct_add_local(m_ret, delta_B);

        vct_copy(cur_point, pre_point); // pre = cur
    }
}

__device__ __forceinline__ void cct345_1_relative_point(float *p, float *pr) {
    float px = p[X];
    float py = p[Y];
    float pz = p[Z];

    // 这两个魔数来自下面向量的相反数
    //Vector3 moving = afterDl2.moveSelf(
    //                directDl2.rotateSelf(BaseUtils.Converter.angleToRadian(-90))
    //                        .changeLengthSelf(secondBend.trajectoryBigRPart2))
    //                .toVector3();
    px -= CCT345_1_MOVE_X;
    py -= CCT345_1_MOVE_Y;

    // 下面是旋转
    // float r_phi = deg2rad(-135);
    // float c = -0.70710678f; // cos(-135) = - cos45
    // float s = -0.70710678f; // sin(-135) = - sin45
    // p[X] = c * x0 - s * y0;
    // p[Y] = s * x0 + c * y0;

    pr[X] = (px - py) * -SIN_45;
    pr[Y] = (px + py) * -COS_45;

    // xz 对称
    pr[Y] *= -1.f;

    // 填上 Z
    pr[Z] = pz;
}

// 函数意义见 cct345_1_relative_point
__device__ __forceinline__ void cct345_1_absolute_m(float *mr, float *m) {
    float mrx = mr[X];
    float mry = mr[Y];
    float mrz = mr[Z];

    // z
    m[Z] = mrz;

    // 对称回去
    mry *= -1.0f;

    // 旋转
    m[X] = (mrx + mry) * -SIN_45;
    m[Y] = (mrx - mry) * SIN_45;

    // 莫名其妙需要全反，我不知道为什么
    vct_dot_a_v(-1.0f, m);

//m[Y] *= -1.f;
//
//        float r_phi = deg2rad(135);
//        float c = cosf(r_phi); // -sin45
//        float s = sinf(r_phi); // sin45
//
//        float x0 = m[X];
//        float y0 = m[Y];
//
//        m[X] = c * x0 - s * y0;
//        m[Y] = s * x0 + c * y0;
//
//        vct_dot_a_v(-1.0f, m);
}

// 粒子走一步 m 磁场，p 位置，v 速度，rm 动质量，sp 速率
__device__  __forceinline__  void particle_run_step(float *m, float *p, float *v, float run_mass, float speed) {
    float a[3]; // 加速度
    float t;    // 运动时间
    float d[3]; // 位置变化 速度变化

    // q v b
    vct_cross(v, m, a); // a = v*b

    vct_dot_a_v(Proton_Charge_Quantity / run_mass, a); // a = q v b / mass 加速度

    t = STEP_RUN / speed; // 运动时长

    vct_dot_a_v_ret(t, v, d); // d = t v 位置变化

    vct_add_local(p, d); // p+=d

    vct_dot_a_v_ret(t, a, d); // d = t a 速度变化

    vct_add_local(v, d); // v+=d
}

/************  TEST *******************/
// 计算整个二极CCT产生的磁场。详见 test_magnet_bicct345_parallel_fill_data 2020年11月15日 测试通过
__global__ void test_magnet_bicct345_parallel(float *data) {
    unsigned int tid = threadIdx.x;

    float p[3] = {0, 0, 0};
    float m_per_wind[3];

    __shared__ float m_total[3];
    if (tid == 0) {
        vct_zero(m_total);
    }
    __syncthreads();

    // 外层
    // Java -0.004067504914360193, 0.006087451294501636, 0.011784791740462989
    // ret -0.004068, 0.006087, 0.011785
//    if (tid > 127 && tid < 256) { // 两层 bi cct

    // 内层
    // Java 0.0031436355039083964, -0.00470478301086915, 0.00888627084434009
    // ret 0.003144, -0.004705, 0.008886
//    if (tid < 128) { // 两层 bi cct

    // Java -9.238694104517966E-4, 0.0013826682836324865, 0.020671062584803078
    // ret -0.000924, 0.001383, 0.020671
    if (tid < 256) {
        dB_cct_wind(
                *((int *) (data + tid * NUMBER_OF_VARIABLES_PER_CCT + 0)), // ksi_deg0
                *(data + tid * NUMBER_OF_VARIABLES_PER_CCT + 10), // start_phi
                *(data + tid * NUMBER_OF_VARIABLES_PER_CCT + 1), // phi0
                data + tid * NUMBER_OF_VARIABLES_PER_CCT + 2, // k
                *(data + tid * NUMBER_OF_VARIABLES_PER_CCT + 6), // a
                *(data + tid * NUMBER_OF_VARIABLES_PER_CCT + 7), // ch_eta0
                *(data + tid * NUMBER_OF_VARIABLES_PER_CCT + 8), // sh_eta0
                p, m_per_wind
        );

        vct_dot_a_v((*(data + tid * NUMBER_OF_VARIABLES_PER_CCT + 9)) * 1e-7f, m_per_wind);

        atomicAdd(&m_total[X], m_per_wind[X]);
        atomicAdd(&m_total[Y], m_per_wind[Y]);
        atomicAdd(&m_total[Z], m_per_wind[Z]);
    }

    __syncthreads();

    if (tid == 0) {
        vct_print(m_total);
    }
}

// 计算整个 AG-CCT 产出的磁场
// 计算整个二极CCT产生的磁场。详见 test_magnet_bicct345_parallel_fill_data 2020年11月15日 测试通过
__global__ void test_magnet_agcct345_parallel(float *data) {
    unsigned int tid = threadIdx.x;

    // Java -0.008153130729662327, 0.12075963610910645, -2.128470062057155
    float p[3] = {0.95, 0.1, 0}; // -0.008154, 0.120739, -2.128648
    float m_per_wind[3];

    __shared__ float m_total[3];
    if (tid == 0) {
        vct_zero(m_total);
    }
    __syncthreads();


    if (tid < 498) {
        dB_cct_wind(
                *((int *) (data + tid * NUMBER_OF_VARIABLES_PER_CCT + 0)), // ksi_deg0
                *(data + tid * NUMBER_OF_VARIABLES_PER_CCT + 10), // start_phi
                *(data + tid * NUMBER_OF_VARIABLES_PER_CCT + 1), // phi0
                data + tid * NUMBER_OF_VARIABLES_PER_CCT + 2, // k
                *(data + tid * NUMBER_OF_VARIABLES_PER_CCT + 6), // a
                *(data + tid * NUMBER_OF_VARIABLES_PER_CCT + 7), // ch_eta0
                *(data + tid * NUMBER_OF_VARIABLES_PER_CCT + 8), // sh_eta0
                p, m_per_wind
        );

        vct_dot_a_v((*(data + tid * NUMBER_OF_VARIABLES_PER_CCT + 9)) * 1e-7f, m_per_wind);

        atomicAdd(&m_total[X], m_per_wind[X]);
        atomicAdd(&m_total[Y], m_per_wind[Y]);
        atomicAdd(&m_total[Z], m_per_wind[Z]);
    }

    __syncthreads();

    if (tid == 0) {
        vct_print(m_total);
    }
}

// 配合 test_magnet_bicct345_parallel 使用，填充参数到 h_data 中
void test_magnet_bicct345_parallel_fill_data(float *h_data) {
    int i;

    int BICCT345_INNER_WIND_NUMBER = 128;
    int BICCT345_OUTER_WIND_NUMBER = 128;
    int AGCCT345_INNER_WIND_NUMBER_0 = 21;
    int AGCCT345_INNER_WIND_NUMBER_1 = 50;
    int AGCCT345_INNER_WIND_NUMBER_2 = 50;
    int AGCCT345_OUTER_WIND_NUMBER_0 = 21;
    int AGCCT345_OUTER_WIND_NUMBER_1 = 50;
    int AGCCT345_OUTER_WIND_NUMBER_2 = 50;

    // bicct
    float bigR = 0.95f;
    float dicct_innerSmallR = 83 * MM + 15 * MM * 2;
    float dicct_outerSmall = 83 * MM + 15 * MM * 3;
    float dicct_bendingAngle_deg = 67.5f;
    float dicct_bendingRadian = dicct_bendingAngle_deg / 180.0f * PI;
    float dicct_tiltAngles[] = {30, 80, 90, 90};
    int dicct_windingNumber = 128;
    float dicct_current = -9664;
    float dicct_phi0 = dicct_bendingRadian / (float) dicct_windingNumber;

    float a, eta0, ch_eta0, sh_eta0, k[TILE_ANGLE_LENGTH];
    int ksi_deg0;

    if (1/*内层 BICCT 便于折叠 2020年11月15日 通过*/) {
        a = sqrtf(bigR * bigR - dicct_innerSmallR * dicct_innerSmallR);
        eta0 = 0.5f * logf((bigR + a) / (bigR - a));
        ch_eta0 = coshf(eta0);
        sh_eta0 = sinhf(eta0);

        k[0] = (1.0f / tanf(dicct_tiltAngles[0] / 180.0f * PI)) / ((float) (0 + 1) * sh_eta0);
        k[1] = (1.0f / tanf(dicct_tiltAngles[1] / 180.0f * PI)) / ((float) (1 + 1) * sh_eta0);
        k[2] = 0.0;
        k[3] = 0.0;

        ksi_deg0 = 0;
        for (i = 0; i < BICCT345_INNER_WIND_NUMBER; i++) { // 0-127
            *((int *) &h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 0]) = ksi_deg0 + 360 * i; // 黑科技

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 1] = dicct_phi0;

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 2] = k[0];
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 3] = k[1];
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 4] = k[2];
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 5] = k[3];

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 6] = a;

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 7] = ch_eta0;
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 8] = sh_eta0;

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 9] = dicct_current;

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 10] = 0.0; // start_phi
        }
    }

    if (1/*外层 BICCT 便于折叠 2020年11月15日 通过*/) {
        dicct_current *= -1; // 电流一定要改，因为让线总是 ksi 正方向，不想 Java 中可以负方向 0~(-endKsi)


        a = sqrtf(bigR * bigR - dicct_outerSmall * dicct_outerSmall);
        eta0 = 0.5f * logf((bigR + a) / (bigR - a));
        ch_eta0 = coshf(eta0);
        sh_eta0 = sinhf(eta0);


        // 以下 5 个参数（4个 k，1 个 phi0）取了相反数，因为要对 ksi-phi 函数做 Y 轴对称
        // Java 代码是 phiKsiFun1 = phiKsiFun1.yAxisSymmetry();
        k[0] = -(1.0f / tanf(dicct_tiltAngles[0] / 180.0f * PI)) / ((float) (0 + 1) * sh_eta0);
        k[1] = -(1.0f / tanf(dicct_tiltAngles[1] / 180.0f * PI)) / ((float) (1 + 1) * sh_eta0);
        k[2] = -0.0;
        k[3] = -0.0;

        dicct_phi0 *= -1;

        ksi_deg0 = -360 * dicct_windingNumber;
        for (; i < BICCT345_INNER_WIND_NUMBER + BICCT345_OUTER_WIND_NUMBER; i++) { // 128-255
            /**
             * 巨大的 bug
             * 不是 ksi_deg0 + 360 * i
             * 而是 ksi_deg0 + 360 * (i - BICCT345_INNER_WIND_NUMBER);
             *
             * 应为 i 不再是从 0 开始了
             */
            *((int *) &h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 0]) =
                    ksi_deg0 + 360 * (i - BICCT345_INNER_WIND_NUMBER);

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 1] = dicct_phi0;

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 2] = k[0];
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 3] = k[1];
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 4] = k[2];
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 5] = k[3];

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 6] = a;

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 7] = ch_eta0;
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 8] = sh_eta0;

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 9] = dicct_current;

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 10] = 0.0; // start_phi
        }
    }

    // agcct
    float agcct_innerSmallR = 83 * MM + 15 * MM * 0;
    float agcct_outerSmall = 83 * MM + 15 * MM * 1;
    float agcct_bendingAngle_degs[3] = {11.716404, 27.93897, 27.844626};
    float agcct_bendingRadians[3] = {agcct_bendingAngle_degs[0] / 180.0f * PI, agcct_bendingAngle_degs[1] / 180.0f * PI,
                                     agcct_bendingAngle_degs[2] / 180.0f * PI};
    float agcct_tiltAngles[] = {90, 30, 90, 90};
    int agcct_windingNumbers[3] = {21, 50, 50};
    float agcct_current = -6000;
    float agcct_phi0s[3] = {
            agcct_bendingRadians[0] / (float) agcct_windingNumbers[0],
            agcct_bendingRadians[1] / (float) agcct_windingNumbers[1],
            agcct_bendingRadians[2] / (float) agcct_windingNumbers[2]
    };

    int agcct_endKsis_deg[3] = {agcct_windingNumbers[0] * 360, agcct_windingNumbers[1] * 360,
                                agcct_windingNumbers[2] * 360};

    if (1/*内层 AGCCT1 便于折叠 2020年11月15日 通过*/) {
        a = sqrtf(bigR * bigR - agcct_innerSmallR * agcct_innerSmallR);
        eta0 = 0.5f * logf((bigR + a) / (bigR - a));
        ch_eta0 = coshf(eta0);
        sh_eta0 = sinhf(eta0);


        k[0] = 0.0;
        k[1] = (1.0f / tanf(agcct_tiltAngles[1] / 180.0f * PI)) / ((float) (1 + 1) * sh_eta0);
        k[2] = 0.0;
        k[3] = 0.0;

        ksi_deg0 = 0;

        for (; i < BICCT345_INNER_WIND_NUMBER + BICCT345_OUTER_WIND_NUMBER +
                   AGCCT345_INNER_WIND_NUMBER_0; i++) { // 256~256+21
            *((int *) &h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 0]) =
                    ksi_deg0 + 360 * (i - BICCT345_INNER_WIND_NUMBER - BICCT345_OUTER_WIND_NUMBER);

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 1] = agcct_phi0s[0];

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 2] = k[0];
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 3] = k[1];
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 4] = k[2];
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 5] = k[3];

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 6] = a;

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 7] = ch_eta0;
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 8] = sh_eta0;

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 9] = agcct_current;

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 10] = 0.0; // start_phi
        }
    }

    if (1/*内层 AGCCT2 便于折叠 2020年11月15日 通过*/) {
        agcct_current *= -1; // 因为反向

        // 反向了，k不反是因为反了两次
        // 第一次：AGCCT 本身交替 BaseUtils.ArrayUtils.dot(tiltAngles, -1)
        // 第二次：因为本 CUDA 代码，只支持正向绕线。原 Java 代码是反向绕线的
        k[0] = 0.0;
        k[1] = (1.0f / tanf(agcct_tiltAngles[1] / 180.0f * PI)) / ((float) (1 + 1) * sh_eta0);
        k[2] = 0.0;
        k[3] = 0.0;

        agcct_phi0s[1] *= -1; // 反向

        ksi_deg0 = -360 * agcct_windingNumbers[1]; // 负数，总是从小到大

        for (; i < BICCT345_INNER_WIND_NUMBER + BICCT345_OUTER_WIND_NUMBER +
                   AGCCT345_INNER_WIND_NUMBER_0 + AGCCT345_INNER_WIND_NUMBER_1; i++) { // 277~327
            *((int *) &h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 0]) = ksi_deg0 + 360 *
                                                                                 (i - BICCT345_INNER_WIND_NUMBER -
                                                                                  BICCT345_OUTER_WIND_NUMBER -
                                                                                  AGCCT345_INNER_WIND_NUMBER_0);

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 1] = agcct_phi0s[1];

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 2] = k[0];
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 3] = k[1];
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 4] = k[2];
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 5] = k[3];

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 6] = a;

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 7] = ch_eta0;
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 8] = sh_eta0;

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 9] = agcct_current;

            // agcct_bendingRadians[0] + agcct_phi0s[0]
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 10] = agcct_bendingRadians[0] + agcct_phi0s[0]; // start_phi
        }

        agcct_current *= -1; // 反回去
        agcct_phi0s[1] *= -1; // 反回去
    }

    if (1/*内层 AGCCT3 便于折叠 2020年11月15日 通过*/) {
        // 不用反向
        k[0] = 0.0;
        k[1] = (1.0f / tanf(agcct_tiltAngles[1] / 180.0f * PI)) / ((float) (1 + 1) * sh_eta0);
        k[2] = 0.0;
        k[3] = 0.0;

        ksi_deg0 = 0;

        for (; i < BICCT345_INNER_WIND_NUMBER + BICCT345_OUTER_WIND_NUMBER +
                   AGCCT345_INNER_WIND_NUMBER_0 + AGCCT345_INNER_WIND_NUMBER_1 +
                   AGCCT345_INNER_WIND_NUMBER_2; i++) { // [327-377)
            *((int *) &h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 0]) = ksi_deg0 + 360 *
                                                                                 (i - BICCT345_INNER_WIND_NUMBER -
                                                                                  BICCT345_OUTER_WIND_NUMBER -
                                                                                  AGCCT345_INNER_WIND_NUMBER_0 -
                                                                                  AGCCT345_INNER_WIND_NUMBER_1);

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 1] = agcct_phi0s[2];

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 2] = k[0];
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 3] = k[1];
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 4] = k[2];
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 5] = k[3];

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 6] = a;

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 7] = ch_eta0;
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 8] = sh_eta0;

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 9] = agcct_current;

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 10] =
                    agcct_bendingRadians[0] + agcct_phi0s[0] + agcct_bendingRadians[1] + agcct_phi0s[1]; // start_phi
        }
    }

    if (1/*外层 AGCCT1 便于折叠 2020年11月15日 通过*/) {
        a = sqrtf(bigR * bigR - agcct_outerSmall * agcct_outerSmall);
        eta0 = 0.5f * logf((bigR + a) / (bigR - a));
        ch_eta0 = coshf(eta0);
        sh_eta0 = sinhf(eta0);

        // 反 .yAxisSymmetry();
        k[0] = -0.0;
        k[1] = -(1.0f / tanf(agcct_tiltAngles[1] / 180.0f * PI)) / ((float) (1 + 1) * sh_eta0);
        k[2] = -0.0;
        k[3] = -0.0;

        agcct_phi0s[0] *= -1; // 反

        ksi_deg0 = -360 * agcct_windingNumbers[0];
        agcct_current *= -1; // 反

        for (; i < BICCT345_INNER_WIND_NUMBER + BICCT345_OUTER_WIND_NUMBER +
                   AGCCT345_INNER_WIND_NUMBER_0 + AGCCT345_INNER_WIND_NUMBER_1 +
                   AGCCT345_INNER_WIND_NUMBER_2 + AGCCT345_OUTER_WIND_NUMBER_0; i++) { // [327-377)
            *((int *) &h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 0]) = ksi_deg0 + 360 *
                                                                                 (i - BICCT345_INNER_WIND_NUMBER -
                                                                                  BICCT345_OUTER_WIND_NUMBER -
                                                                                  AGCCT345_INNER_WIND_NUMBER_0 -
                                                                                  AGCCT345_INNER_WIND_NUMBER_1 -
                                                                                  AGCCT345_INNER_WIND_NUMBER_2);

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 1] = agcct_phi0s[0];

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 2] = k[0];
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 3] = k[1];
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 4] = k[2];
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 5] = k[3];

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 6] = a;

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 7] = ch_eta0;
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 8] = sh_eta0;

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 9] = agcct_current;

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 10] = 0.0; // start_phi
        }

        agcct_phi0s[0] *= -1; // 反回去
        agcct_current *= -1; // 反回去
    }

    if (1/*外层 AGCCT2 便于折叠 2020年11月15日 通过*/) {

        // 反 BaseUtils.ArrayUtils.dot(tiltAngles, -1)
        k[0] = -0.0;
        k[1] = -(1.0f / tanf(agcct_tiltAngles[1] / 180.0f * PI)) / ((float) (1 + 1) * sh_eta0);
        k[2] = -0.0;
        k[3] = -0.0;

        ksi_deg0 = 0;

        for (; i < BICCT345_INNER_WIND_NUMBER + BICCT345_OUTER_WIND_NUMBER +
                   AGCCT345_INNER_WIND_NUMBER_0 + AGCCT345_INNER_WIND_NUMBER_1 +
                   AGCCT345_INNER_WIND_NUMBER_2 + AGCCT345_OUTER_WIND_NUMBER_0 +
                   AGCCT345_OUTER_WIND_NUMBER_1; i++) { // 398 - 398+50
            *((int *) &h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 0]) = ksi_deg0 + 360 *
                                                                                 (i - BICCT345_INNER_WIND_NUMBER -
                                                                                  BICCT345_OUTER_WIND_NUMBER -
                                                                                  AGCCT345_INNER_WIND_NUMBER_0 -
                                                                                  AGCCT345_INNER_WIND_NUMBER_1 -
                                                                                  AGCCT345_INNER_WIND_NUMBER_2 -
                                                                                  AGCCT345_OUTER_WIND_NUMBER_0);

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 1] = agcct_phi0s[1];

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 2] = k[0];
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 3] = k[1];
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 4] = k[2];
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 5] = k[3];

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 6] = a;

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 7] = ch_eta0;
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 8] = sh_eta0;

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 9] = agcct_current;

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 10] = agcct_bendingRadians[0] + agcct_phi0s[0]; // start_phi
        }
    }

    if (1/*外层 AGCCT3 便于折叠 2020年11月15日 通过*/) {

        // .yAxisSymmetry();
        k[0] = -0.0;
        k[1] = -(1.0f / tanf(agcct_tiltAngles[1] / 180.0f * PI)) / ((float) (1 + 1) * sh_eta0);
        k[2] = -0.0;
        k[3] = -0.0;

        agcct_current *= -1;//反
        agcct_phi0s[2] *= -1;//反

        ksi_deg0 = -360 * agcct_windingNumbers[2];

        for (; i < BICCT345_INNER_WIND_NUMBER + BICCT345_OUTER_WIND_NUMBER +
                   AGCCT345_INNER_WIND_NUMBER_0 + AGCCT345_INNER_WIND_NUMBER_1 +
                   AGCCT345_INNER_WIND_NUMBER_2 + AGCCT345_OUTER_WIND_NUMBER_0 +
                   AGCCT345_OUTER_WIND_NUMBER_1 + AGCCT345_OUTER_WIND_NUMBER_2; i++) { // 348 - 398
            *((int *) &h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 0]) = ksi_deg0 + 360 *
                                                                                 (i - BICCT345_INNER_WIND_NUMBER -
                                                                                  BICCT345_OUTER_WIND_NUMBER -
                                                                                  AGCCT345_INNER_WIND_NUMBER_0 -
                                                                                  AGCCT345_INNER_WIND_NUMBER_1 -
                                                                                  AGCCT345_INNER_WIND_NUMBER_2 -
                                                                                  AGCCT345_OUTER_WIND_NUMBER_0 -
                                                                                  AGCCT345_OUTER_WIND_NUMBER_1);

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 1] = agcct_phi0s[2];

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 2] = k[0];
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 3] = k[1];
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 4] = k[2];
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 5] = k[3];

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 6] = a;

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 7] = ch_eta0;
            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 8] = sh_eta0;

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 9] = agcct_current;

            h_data[i * NUMBER_OF_VARIABLES_PER_CCT + 10] =
                    agcct_bendingRadians[0] + agcct_phi0s[0] + agcct_bendingRadians[1] + agcct_phi0s[1]; // start_phi
        }

        agcct_current *= -1; //反回去
        agcct_phi0s[2] *= -1; //反回去
    }
}

int main() {
    unsigned int blockNumber = 1;
    unsigned int threadNumber = 1024; // 最大CCT匝数和。当前 (128*2+(21+50+50)*2)*2)


    float *h_data; // 内存
    float *d_data; // 显存
    unsigned int data_size = blockNumber * threadNumber * NUMBER_OF_VARIABLES_PER_CCT * sizeof(float);

    h_data = (float *) malloc(data_size);
    hipMalloc((void **) &d_data, data_size);

    test_magnet_bicct345_parallel_fill_data(h_data);

    hipMemcpy(d_data, h_data, data_size, hipMemcpyHostToDevice);

    test_magnet_agcct345_parallel<<<blockNumber, threadNumber >>>(d_data);

    hipMemcpy(h_data, d_data, data_size, hipMemcpyDeviceToHost);

    printf("hello -- host\n");

    free(h_data);
    hipFree(d_data);

    return 0;
}

